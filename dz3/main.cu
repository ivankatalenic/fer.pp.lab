#include "hip/hip_runtime.h"
#include <iostream>

#include <hip/hip_runtime.h>

// Complex pointwise multiplication and scale
static __global__ void ComplexPointwiseMulAndScale(
	hipfftComplex* arr,
	int len,
	float scale,
	int offset
) {
	const unsigned i{blockIdx.x * blockDim.x + threadIdx.x};
	if (i >= len) {
		return;
	}
	arr[i + offset] = ComplexScale(
		ComplexMul(arr[i + offset], arr[i % offset]),
		scale
	);
}

int main(int argc, char* argv[]) {
	return 0;
}
